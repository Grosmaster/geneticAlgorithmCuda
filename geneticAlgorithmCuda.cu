#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
//#include "pch.h"
#include <iostream>
using namespace std;
#include <stdio.h>
#include <algorithm>
#include <ctime>
#include <cstdlib> // Для работы с функцией system()
/*
const int sizePoint = 5;
const int sizeIndividum = 5;
const int mathValueMutation = 5;
const float dispersionMutation = 5.0f;
const int powCount = 3;
const float randMaxCount = 20.0f;
*/

const int sizePoint = 500;
const int sizeIndividum = 1000;
const int mathValueMutation = 5;
const float dispersionMutation = 5.0f;
const int powCount = 3;
const float randMaxCount = 20.0f;
const int maxPokoleney = 30;

__global__ void errorsKernel(float *points, float *individs, float *errors, int powCount, int sizePoint)
{

	int id = threadIdx.x;
	float ans = 0;
	int x = 1;
	for (int i = 0; i < sizePoint; i++)
	{
		for (int j = 0; j < powCount; j++)
		{
			for (int k = 0; k < j; k++)
			{
				x *= i;
			}
			x *= individs[id*powCount + j];
			ans += x;
			x = 1;
		}

		ans = points[i] - ans;
		errors[id] += sqrt(ans * ans);
		ans = 0;
	}
}


void testErrorsKernel(float* points, float* individs, float* errors, int powCount, int sizePoint, int sizeIndividum)
{
	for (int id = 0; id < sizeIndividum; id++)
	{
		float ans = 0.0f;
		errors[id] = 0.0f;
		int x = 0;
		for (int i = 0; i < sizePoint; i++)
		{
			for (int j = 0; j < powCount; j++)
			{
				x = pow(i, j);
				x *= individs[id*powCount + j];
				ans += x;
				x = 0;
			}

			ans = points[i] - ans;
			errors[id] += sqrt(ans * ans);
			ans = 0;
		}
	}
}

float RandomFloat(float a, float b) {
	float random = ((float)rand()) / (float)RAND_MAX;
	float diff = b - a;
	float r = random * diff;
	return a + r;
}

void cpu() {
	float* pointsH = new float[sizePoint];
	for (int i = 0; i < sizePoint; i++)
	{
		pointsH[i] = RandomFloat(0, 20);
	}

	float* individumsH = new float[sizeIndividum * powCount];
	for (int i = 0; i < sizeIndividum * powCount; i++)
	{
		individumsH[i] = RandomFloat(0, randMaxCount);
	}

	float* errorsH = new float[sizeIndividum];
	for (int i = 0; i < sizeIndividum; i++)
	{
		errorsH[i] = 1000;
	}

	unsigned int start_time = clock(); // начальное время

	for (int pokolenie = 0; pokolenie < maxPokoleney; pokolenie++)
	{
		testErrorsKernel(pointsH, individumsH, errorsH, powCount, sizePoint, sizeIndividum);

		float* errorsCrossOver = new float[sizeIndividum];

		for (size_t i = 0; i != sizeIndividum; ++i)
			errorsCrossOver[i] = errorsH[i];
		sort(errorsCrossOver, errorsCrossOver + sizeIndividum);

		int merodianCrossOvering = sizeIndividum / 2;
		float merodianErrorCrossOvering = errorsCrossOver[merodianCrossOvering];
		float* theBestInd = new float[powCount];

		for (size_t i = 0; i < sizeIndividum; i++)
		{
			if (merodianErrorCrossOvering < errorsH[i]) {
				for (size_t j = 0; j < powCount; j++)
				{
					individumsH[i * powCount + j] = 0;
				}
			}
			if (errorsH[i] == errorsCrossOver[0]) {
				for (int j = 0; j < powCount; j++)
				{
					theBestInd[j] = individumsH[i *  powCount + j];
				}
			}
		}

		printf("error = %f\n", errorsCrossOver[0]);

		for (int i = 0; i < sizeIndividum * powCount; i++)
		{
			if (individumsH[i] == 0) {
				individumsH[i] = theBestInd[rand() % powCount];
			}

			if (mathValueMutation >(rand() % 100 + 1)) {
				individumsH[i] += RandomFloat(-dispersionMutation, dispersionMutation);
			}
		}
	}
	unsigned int end_time = clock(); // конечное время
	unsigned int search_time = end_time - start_time; // искомое время
	printf("search_time_cpu = %i\n", search_time);
}

void gpu() {
	float* pointsH = new float[sizePoint];
	for (int i = 0; i < sizePoint; i++)
	{
		pointsH[i] = RandomFloat(0, 20);
	}

	float* individumsH = new float[sizeIndividum * powCount];
	for (int i = 0; i < sizeIndividum * powCount; i++)
	{
		individumsH[i] = RandomFloat(0, randMaxCount);
	}

	float* errorsH = new float[sizeIndividum];
	for (int i = 0; i < sizeIndividum; i++)
	{
		errorsH[i] = 1000;
	}

	unsigned int start_time_gpu = clock(); // начальное время
	float* pointsD = NULL;
	float* individumsD = NULL;
	float* errorsD = NULL;

	for (int pokolenie = 0; pokolenie < maxPokoleney; pokolenie++)
	{

		int sizeIndividumBytes = sizeIndividum * powCount * sizeof(float);
		int sizePointBytes = sizePoint * sizeof(float);

		hipMalloc((void**)&pointsD, sizePointBytes);
		hipMalloc((void**)&individumsD, sizeIndividumBytes*powCount);
		hipMalloc((void**)&errorsD, sizeIndividum * sizeof(float));

		hipMemcpy(pointsD, pointsH, sizePointBytes, hipMemcpyHostToDevice);
		hipMemcpy(individumsD, individumsH, sizeIndividumBytes, hipMemcpyHostToDevice);
		hipMemcpy(errorsD, errorsH, sizeIndividumBytes, hipMemcpyHostToDevice);

		errorsKernel << <1, sizeIndividum >> > (pointsD, individumsD, errorsD, powCount, sizePoint);

		hipMemcpy(errorsH, errorsD, sizeIndividum * sizeof(float), hipMemcpyDeviceToHost);

		//----------------------
		float* errorsCrossOver = new float[sizeIndividum];

		for (size_t i = 0; i != sizeIndividum; ++i)
			errorsCrossOver[i] = errorsH[i];
		sort(errorsCrossOver, errorsCrossOver + sizeIndividum);
		printf("error = %f\n", errorsCrossOver[0]);
		int merodianCrossOvering = sizeIndividum / 2;
		float merodianErrorCrossOvering = errorsCrossOver[merodianCrossOvering];
		float* theBestInd = new float[powCount];

		for (size_t i = 0; i < sizeIndividum; i++)
		{
			if (merodianErrorCrossOvering < errorsH[i]) {
				for (size_t j = 0; j < powCount; j++)
				{
					individumsH[i * powCount + j] = 0;
				}
			}
			if (errorsH[i] == errorsCrossOver[0]) {
				for (int j = 0; j < powCount; j++)
				{
					theBestInd[j] = individumsH[i *  powCount + j];
				}
			}
		}

		for (int i = 0; i < sizeIndividum * powCount; i++)
		{
			if (individumsH[i] == 0) {
				individumsH[i] = theBestInd[rand() % powCount];
			}

			if (mathValueMutation >(rand() % 100 + 1)) {
				individumsH[i] += RandomFloat(-dispersionMutation, dispersionMutation);
			}
		}
	}
	unsigned int end_time_gpu = clock(); // конечное время
	unsigned int search_time_gpu = end_time_gpu - start_time_gpu; // искомое время

	printf("search_time_gpu = %i\n", search_time_gpu);

	hipFree(pointsD);
	hipFree(individumsD);
	hipFree(errorsD);

	delete pointsH;
	delete individumsH;
	delete errorsH;
}

int main()
{
	cpu();
	gpu();
	system("pause");
	return 0;
}
